
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

// setting GPU device
const int device = 0;
#define BLOCK_SIZE 16

void __global__ matvec(double *y, double *A, double *x, int M, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < M && j < N) atomicAdd(&y[i], A[i * N + j] * x[j]);
}

int main(int argc, char *argv[]) {

    // warm up:
    hipSetDevice(device);
    double *dummy_d;
    hipMalloc((void**)&dummy_d, 0);

    // command line argument sets the dimensions of the image
    int M,N;
    if ( argc == 3 ) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
    } else {
        // default   
        N = 2048;
        M = 2048;
    }

    double *d_A, *d_b, *d_c;
    double *h_A, *h_b, *h_c;
    int size_A = sizeof(double)*N*M, size_b = sizeof(double)*N, size_c = sizeof(double)*M;

    // GPU
    // Allocate memory on host and device
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_b, size_b);
    hipMalloc((void**)&d_c, size_c);
    hipHostMalloc((void**)&h_A, size_A, hipHostMallocDefault);
    hipHostMalloc((void**)&h_b, size_b, hipHostMallocDefault);
    hipHostMalloc((void**)&h_c, size_c, hipHostMallocDefault);


    // initialize d_A and d_b
    double init_A = 2.0, init_b = 2.0;
    // double check_ele = (double)M*(init_A * (double)N + init_b * (double)N);
    for (int i = 0; i < M*N; i++) h_A[i] = init_A;
    for (int i = 0; i < N; i++) h_b[i] = init_b;

    // copy data
    double time,time_end,time_IO_1,time_IO_2,time_compute,time_compute_end,tot_time_compute;
    time = omp_get_wtime();
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    time_IO_1 = omp_get_wtime()- time;
    
    // define grid and threads/block
    dim3 dim_grid(((M+BLOCK_SIZE-1) / BLOCK_SIZE), ((N+BLOCK_SIZE-1) / BLOCK_SIZE));
    dim3 dim_block(BLOCK_SIZE,BLOCK_SIZE);
    time_compute = omp_get_wtime();   
    matvec<<<dim_grid, dim_block>>>(d_c, d_A, d_b, M, N);
    hipDeviceSynchronize();
    time_compute_end = omp_get_wtime();
    
    // Copy result back to host
    hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);
    time_end = omp_get_wtime();
    time_IO_2 = time_end - time_compute_end;
    tot_time_compute = time_compute_end - time_compute;

    // stats
    double GB = 1.0e-09;
    double gflops  = (N * M * 2 / tot_time_compute) * GB;
    double memory  = size_A + size_b + size_c;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    printf("%g\t", gflops / 70.65); // pct. Gflops

    printf("%g\t", memoryGBs); // bandwidth GB/s
    printf("%g\t", memoryGBs / 8.98); // pct. bandwidth GB/s

    printf("%g\t", time_end - time); // total time
    printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    printf("%g\n", tot_time_compute); // compute time

    // Cleanup
    hipHostFree(h_A), hipHostFree(h_b), hipHostFree(h_c); 
    hipFree(d_A), hipFree(d_b), hipFree(d_c); 

    return(0);
}
