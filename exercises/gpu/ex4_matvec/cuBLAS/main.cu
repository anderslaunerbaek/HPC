
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hipblas.h>

// setting GPU device
const int device = 0;
#define BLOCK_SIZE 16

void matvec(hipblasHandle_t handle, double *y, double *A, double *x, int M, int N) {

    int lda = M; // leading dimension of A
    double alpha = 1.0, beta = 0.0; // constants for dgemv
    int incx = 1, incy = 1;

    hipblasDgemv(handle, HIPBLAS_OP_T, M, N, &alpha, A, lda, x, incx, &beta, y, incy);
}


int main(int argc, char *argv[]) {
    // cuBLAS handle??
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    // initialization of CUBLAS
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
      printf ("CUBLAS initialization failed\n");
      return EXIT_FAILURE;
    }


    // warm up:
    hipSetDevice(device);
    double *dummy_d;
    hipMalloc((void**)&dummy_d, 0);

    // command line argument sets the dimensions of the image
    int M,N;
    if ( argc == 3 ) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
    } else {
        // default   
        N = 2048;
        M = 2048;
    }

    double *d_A, *d_b, *d_c;
    double *h_A, *h_b, *h_c;
    int size_A = sizeof(double)*N*M, size_b = sizeof(double)*N, size_c = sizeof(double)*M;

    // GPU
    // Allocate memory on host and device
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_b, size_b);
    hipMalloc((void**)&d_c, size_c);
    hipHostMalloc((void**)&h_A, size_A, hipHostMallocDefault);
    hipHostMalloc((void**)&h_b, size_b, hipHostMallocDefault);
    hipHostMalloc((void**)&h_c, size_c, hipHostMallocDefault);


    // initialize d_A and d_b
    double init_A = 2.0, init_b = 2.0;
    // double check_ele = (double)M*(init_A * (double)N + init_b * (double)N);
    for (int i = 0; i < M*N; i++) h_A[i] = init_A;
    for (int i = 0; i < N; i++) h_b[i] = init_b;

    // copy data
    double time,time_end,time_IO_1,time_IO_2,time_compute,time_compute_end,tot_time_compute;
    time = omp_get_wtime();
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    time_IO_1 = omp_get_wtime()- time;
    
   
    time_compute = omp_get_wtime();   
    matvec(handle, d_c, d_A, d_b, M, N);
    hipDeviceSynchronize();
    time_compute_end = omp_get_wtime();
    
    // Copy result back to host
    hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);
    time_end = omp_get_wtime();
    time_IO_2 = time_end - time_compute_end;
    tot_time_compute = time_compute_end - time_compute;

    // stats
    double GB = 1.0e-09;
    double gflops  = (N * M * 2 / tot_time_compute) * GB;
    double memory  = size_A + size_b + size_c;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    printf("%g\t", gflops / 70.65); // pct. Gflops

    printf("%g\t", memoryGBs); // bandwidth GB/s
    printf("%g\t", memoryGBs / 8.98); // pct. bandwidth GB/s

    printf("%g\t", time_end - time); // total time
    printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    printf("%g\n", tot_time_compute); // compute time

    // Cleanup
    hipHostFree(h_A), hipHostFree(h_b), hipHostFree(h_c); 
    hipFree(d_A), hipFree(d_b), hipFree(d_c); 
    hipblasDestroy(handle);

    return(0);
}
