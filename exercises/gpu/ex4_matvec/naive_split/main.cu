
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

// setting GPU device
const int device0 = 0;
const int device1 = 1;
#define BLOCK_SIZE 16

void __global__ matvec(double *y, double *A, double *x, int M, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < M && j < N) atomicAdd(&y[i], A[i * N + j] * x[j]);
}

int main(int argc, char *argv[]) {

    // warm up:
    double *dummy_d;
    hipSetDevice(device0);
    hipMalloc((void**)&dummy_d, 0);
    hipSetDevice(device1);
    hipMalloc((void**)&dummy_d, 0);

    // command line argument sets the dimensions of the image
    int M,N;
    if ( argc == 3 ) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
    } else {
        // default   
        N = 2048;
        M = 2048;
    }

    double *d0_A, *d0_b, *d0_c, *d1_A, *d1_b, *d1_c;
    double *h_A, *h_b, *h_c;
    int size_A = sizeof(double)*N*M, size_b = sizeof(double)*N, size_c = sizeof(double)*M;

    // GPU MULTI
    // Allocate memory on host and device
    hipSetDevice(device0);
    hipMalloc((void**)&d0_A, size_A/2);
    hipMalloc((void**)&d0_b, size_b);
    hipMalloc((void**)&d0_c, size_c/2);
    hipSetDevice(device1);
    hipMalloc((void**)&d1_A, size_A/2);
    hipMalloc((void**)&d1_b, size_b);
    hipMalloc((void**)&d1_c, size_c/2);
    
    hipHostMalloc((void**)&h_A, size_A, hipHostMallocDefault);
    hipHostMalloc((void**)&h_b, size_b, hipHostMallocDefault);
    hipHostMalloc((void**)&h_c, size_c, hipHostMallocDefault);




    // initialize d_A and d_b
    double init_A = 2.0, init_b = 2.0;
    // double check_ele = (double)M*(init_A * (double)N + init_b * (double)N);
    for (int i = 0; i < M*N; i++) h_A[i] = init_A;
    for (int i = 0; i < N; i++) h_b[i] = init_b;


    // copy data
    double time,time_end,time_IO_1,time_IO_2,time_compute,time_compute_end,tot_time_compute;
    time = omp_get_wtime();
    hipSetDevice(device0);
    hipMemcpy(d0_A, h_A, size_A/2, hipMemcpyHostToDevice);
    hipMemcpy(d0_b, h_b, size_b, hipMemcpyHostToDevice);
    
    hipSetDevice(device1);
    hipMemcpy(d1_A, h_A + size_A/2, size_A/2, hipMemcpyHostToDevice);
    hipMemcpy(d1_b, h_b, size_b, hipMemcpyHostToDevice);
    time_IO_1 = omp_get_wtime()- time;
    
    // define grid and threads/block
    dim3 dim_grid((((M/2)+BLOCK_SIZE-1) / BLOCK_SIZE), (((N/2)+BLOCK_SIZE-1) / BLOCK_SIZE));
    dim3 dim_block(BLOCK_SIZE,BLOCK_SIZE);
    time_compute = omp_get_wtime();   
    hipSetDevice(device0);
    matvec<<<dim_grid, dim_block>>>(d0_c, d0_A, d0_b, M/2, N);
    hipSetDevice(device1);
    matvec<<<dim_grid, dim_block>>>(d1_c, d1_A, d1_b, M/2, N);
    hipDeviceSynchronize();
    hipSetDevice(device0);
    hipDeviceSynchronize();
    time_compute_end = omp_get_wtime();
    
    // Copy result back to host
    hipSetDevice(device0);
    hipMemcpy(h_c, d0_c, size_c/2, hipMemcpyDeviceToHost);
    hipSetDevice(device1);
    hipMemcpy(h_c + size_c/2, d1_c, size_c/2, hipMemcpyDeviceToHost);

    time_end = omp_get_wtime();
    time_IO_2 = time_end - time_compute_end;
    tot_time_compute = time_compute_end - time_compute;

    // stats
    double GB = 1.0e-09;
    double gflops  = (N * M * 2 / tot_time_compute) * GB;
    double memory  = size_A + size_b + size_c;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    printf("%g\t", gflops / 141.30); // pct. Gflops

    printf("%lg\t", memoryGBs); // bandwidth GB/s
    printf("%lg\t", memoryGBs / 17.96); // pct. bandwidth GB/s

    printf("%g\t", time_end - time); // total time
    printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    printf("%g\n", tot_time_compute); // compute time


    // Cleanup
    hipHostFree(h_A), hipHostFree(h_b), hipHostFree(h_c); 
    hipFree(d0_A), hipFree(d0_b), hipFree(d0_c); 
    hipFree(d1_A), hipFree(d1_b), hipFree(d1_c); 

    return(0);
}


