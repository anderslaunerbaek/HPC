
#include <hip/hip_runtime.h>
void __global__ mandel_gpu(int disp_width, int disp_height, int *image, int max_iter, int blk_size) {

    double  scale_real, scale_imag;
    double  x, y, u, v, u2, v2;

    scale_real = 3.5 / (double)disp_width;
    scale_imag = 3.5 / (double)disp_height;

    int j = blockIdx.y * blockDim.y + threadIdx.y;  // WIDTH
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // HEIGHT
    int idx = i * disp_height + j;
    
    // check if inside picture
    if (i >= disp_width || j >= disp_height) return;

    //
    x = ((double)i * scale_real) - 2.25;
    y = ((double)j * scale_imag) - 1.75;

    u    = 0.0;
    v    = 0.0;
    u2   = 0.0;
    v2   = 0.0;
    int iter = 0;

    while ( u2 + v2 < 4.0 &&  iter < max_iter ) {
        v = 2 * v * u + y;
        u = u2 - v2 + x;
        u2 = u*u;
        v2 = v*v;
        iter ++;
    }

    // if we exceed max_iter, reset to zero
    iter = iter == max_iter ? 0 : iter;

    image[idx] = iter;  
}

