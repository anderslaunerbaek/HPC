#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include "mandelgpu.h"
#include "writepng.h"

// setting GPU device
const int device = 0;

int main(int argc, char *argv[]) {

    int width, height, max_iter, *h_image_gpu, *d_image_gpu;

    width    = 10000;
    height   = 10000;
    max_iter = 400;

    // warm up:
    hipSetDevice(device);
    double *dummy_d;
    hipMalloc((void**)&dummy_d, 0);
    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);


    // GPU
    // Alloc mem on host and device
    hipHostMalloc((void **)&h_image_gpu, width * height * sizeof(int));
    hipMalloc((void **)&d_image_gpu, width * height * sizeof(int));

    //   
    int bs = 16;
    dim3 block_size(bs, bs);
    dim3 grid_size(width / block_size.x, height / block_size.y);
    
    double time = omp_get_wtime();
    mandel_gpu<<<grid_size, block_size>>>(width, height, d_image_gpu, max_iter, bs);
    //mandel_gpu<<<1,1>>>(width, height, d_image_gpu, max_iter, 1);
    hipDeviceSynchronize();
    double time_compute = omp_get_wtime();

    // Copy result back to host
    hipMemcpy(h_image_gpu, d_image_gpu, width * height * sizeof(int), hipMemcpyDeviceToHost);
    double time_IO = omp_get_wtime();

    printf("Compute = %3.2f seconds\n", time_compute - time);
    printf("IO =      %3.2f seconds\n", time_IO - time_compute);
    printf("Total =   %3.2f seconds\n", time_IO - time);

    writepng("mandelbrotgpu.png", h_image_gpu, width, height);

    // Cleanup
    hipHostFree(h_image_gpu); 
    hipFree(d_image_gpu);

    return(0);
}

