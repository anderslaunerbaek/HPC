
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>

const int device = 0;

int main(int argc, char *argv[])
{
    // Wake up GPU from power save state.
    printf("Warming up device %i ... ", device); fflush(stdout);
    double time = omp_get_wtime();
    hipSetDevice(device);           // Set the device to 0 or 1.
    double *dummy_d;
    hipMalloc((void**)&dummy_d, 0); // We force the creation of context on the
                                     // device by allocating a dummy variable.
    printf("time = %3.2f seconds\n", omp_get_wtime() - time);
}
