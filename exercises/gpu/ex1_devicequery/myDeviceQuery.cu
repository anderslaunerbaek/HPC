#include <stdio.h>
#include <hip/hip_runtime_api.h>

int main(int argc, char *argv[])
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int device = 0; device < nDevices; device++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);
        printf("Device %i: \"%s\".\n", device, prop.name);
        printf("  Multiprocessors: %i\n", prop.multiProcessorCount);
        printf("  Cores: %i\n",_ConvertSMVer2Cores(prop.major, prop.minor)
               * prop.multiProcessorCount);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Maximum number of threads per block: %d.\n",
               prop.maxThreadsPerBlock);
    }
}
