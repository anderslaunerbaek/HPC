
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>

const int device = 0;

// kernel function
__global__ void my_kernel() {
	//
	int thread_i = threadIdx.x;
	int thread_max = blockDim.x;
	int block_i = blockIdx.x;

	int glo_thread_i = blockDim.x * blockIdx.x + threadIdx.x;
	int glo_thread_i_max = gridDim.x * blockDim.x;

	printf("Hello world! I'm thread %i out of %i in block %i. My global thread id is %i out of %i.\n", thread_i,thread_max,block_i,glo_thread_i,glo_thread_i_max);
}


int main(int argc, char *argv[]) {
	// Wake up GPU from power save state.
	printf("Warming up device %i ... ", device); fflush(stdout);
	double time = omp_get_wtime();
	hipSetDevice(device);           // Set the device to 0 or 1.
	double *dummy_d;
	hipMalloc((void**)&dummy_d, 0); // We force the creation of context on the
	// device by allocating a dummy variable.
	printf("time = %3.2f seconds\n", omp_get_wtime() - time);

	// program 
	int n_blk, n_threads;

	if (argc == 3 ) {
		n_blk = atoi(argv[1]);
		n_threads = atoi(argv[2]);
	}
	else {
		// use default N
		n_blk = 1;
		n_threads = 32;
	}

	//

	printf("n_blk  %i ; n_threads %i\n",n_blk, n_threads);

	my_kernel<<<n_blk,n_threads>>>();
	hipDeviceSynchronize();
}