extern "C" { 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

void write_result(double *U, int N, double delta, char filename[40]) {
    double u, y, x;
    FILE *matrix=fopen(filename, "w");
    for (int i = 0; i < N; i++) {
        x = -1.0 + i * delta + delta * 0.5;
        for (int j = 0; j < N; j++) {
            y = -1.0 + j * delta + delta * 0.5;
            u = U[i*N + j];
            fprintf(matrix, "%g\t%g\t%g\n", x,y,u);
        }
    }
    fclose(matrix);
}
}

const int device0 = 0;
const int device1 = 1;
#define BLOCK_SIZE 16


void __global__ jac_gpu3_d0(int N, double delta, int max_iter, double *f, double *u, double *u_old, double *d1_u_old) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < (N/2-1) && j < (N-1) && i > 0 && j > 0) {
        u[i*N + j] = 0.25 * (u_old[(i-1)*N + j] + u_old[(i+1)*N + j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] + delta*delta*f[i*N + j]);    
    }
    else if (i == (N/2-1) && j < (N-1) && j > 0) {
        u[i*N + j] = 0.25 * (u_old[(i-1)*N + j] + d1_u_old[j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] + delta*delta*f[i*N + j]);    
    }
}

void __global__ jac_gpu3_d1(int N, double delta, int max_iter, double *f, double *u, double *u_old, double *d0_u_old) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < (N/2-1) && j < (N-1) && i > 0 && j > 0) { // i < N/2
        u[i*N + j] = 0.25 * (u_old[(i-1)*N + j] + u_old[(i+1)*N + j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] + delta*delta*f[i*N + j]);        
    }
    else if (i == 0 && j < (N-1) && j > 0) {
        u[i*N + j] = 0.25 * (d0_u_old[(N/2-1)*N + j] + u_old[(i+1)*N+j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] + delta*delta*f[i*N + j]);    
    }
}

int main(int argc, char *argv[]) {

    // warm up:
    double *dummy_d;
    hipSetDevice(device0);
    hipMalloc((void**)&dummy_d, 0);
    hipSetDevice(device1);
    hipMalloc((void**)&dummy_d, 0);

    int max_iter, N,i,j;

    if (argc == 3) {
        N = atoi(argv[1]) + 2;
        max_iter = atoi(argv[2]);
    }
    else {
        // use default N
        N = 128 + 2;
        max_iter = 5000;
    }
    double delta = 2.0/N;

    // allocate mem
    double *h_f, *h_u, *h_u_old;
    double *d0_f, *d0_u, *d0_u_old, *d1_f, *d1_u, *d1_u_old;

    int size_f = N * N * sizeof(double);
    int size_u = N * N * sizeof(double);
    int size_u_old = N * N * sizeof(double);
    int size_f_p2 = N*N/2;
    int size_u_p2 = N*N/2;
    int size_u_old_p2 = N*N/2;

    //Allocate memory on device
    hipSetDevice(device0);
    hipMalloc((void**)&d0_f, size_f/2);
    hipMalloc((void**)&d0_u, size_u/2);
    hipMalloc((void**)&d0_u_old, size_u_old/2);
    hipSetDevice(device1);
    hipMalloc((void**)&d1_f, size_f/2);
    hipMalloc((void**)&d1_u, size_u/2);
    hipMalloc((void**)&d1_u_old, size_u_old/2);
    //Allocate memory on host
    hipHostMalloc((void**)&h_f, size_f, hipHostMallocDefault);
    hipHostMalloc((void**)&h_u, size_u, hipHostMallocDefault);
    hipHostMalloc((void**)&h_u_old, size_u_old, hipHostMallocDefault);

    // initialize boarder
    for (i = 0; i < N; i++){
        for (j = 0; j < N; j++){
            if (i >= N * 0.5  &&  i <= N * 2.0/3.0  &&  j >= N * 1.0/6.0  &&  j <= N * 1.0/3.0)
                h_f[i*N + j] = 200.0;
            else
                h_f[i*N + j] = 0.0; 

            if (i == (N - 1) || i == 0 || j == (N - 1)){
                h_u[i*N + j] = 20.0;
                h_u_old[i*N + j] = 20.0;
            }
            else{
                h_u[i*N + j] = 0.0;
                h_u_old[i*N + j] = 0.0;
            } 
        }
    }
    
    //Copy memory host -> device
    double time_tmp = omp_get_wtime(); 
    hipSetDevice(device0);
    hipMemcpy(d0_f, h_f, size_f/2, hipMemcpyHostToDevice);
    hipMemcpy(d0_u, h_u, size_u/2, hipMemcpyHostToDevice);
    hipMemcpy(d0_u_old, h_u_old, size_u_old/2, hipMemcpyHostToDevice);
    hipSetDevice(device1);
    hipMemcpy(d1_f, h_f + size_f_p2, size_f/2, hipMemcpyHostToDevice);
    hipMemcpy(d1_u, h_u + size_u_p2, size_u/2, hipMemcpyHostToDevice);
    hipMemcpy(d1_u_old, h_u_old + size_u_old_p2, size_u_old/2, hipMemcpyHostToDevice);
    double time_IO_1 = omp_get_wtime() - time_tmp; 

    // peer enable
    hipSetDevice(device0);
    hipDeviceEnablePeerAccess(device1,0);
    hipSetDevice(device1);
    hipDeviceEnablePeerAccess(device0,0);

    // do program
    int k = 0;
    dim3 dim_grid(((N +BLOCK_SIZE-1) / BLOCK_SIZE), ((N/2+BLOCK_SIZE-1) / BLOCK_SIZE));
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    double *temp_p;
    double time_compute = omp_get_wtime(); 
    while (k < max_iter) {
        // Set u_old = u device 0
        temp_p = d0_u;
        d0_u = d0_u_old;
        d0_u_old = temp_p;
        // Set u_old = u device 0
        temp_p = d1_u;
        d1_u = d1_u_old;
        d1_u_old = temp_p;

        hipSetDevice(device0);
        jac_gpu3_d0<<<dim_grid, dim_block>>>(N, delta, max_iter, d0_f, d0_u, d0_u_old, d1_u_old);
        hipSetDevice(device1);
        jac_gpu3_d1<<<dim_grid, dim_block>>>(N, delta, max_iter, d1_f, d1_u, d1_u_old, d0_u_old);
        hipDeviceSynchronize();
        hipSetDevice(device0);
        hipDeviceSynchronize();
        k++;
    }/* end while */
    double tot_time_compute = omp_get_wtime() - time_compute;
    // end program

    //Copy memory host -> device
    time_tmp = omp_get_wtime(); 
    hipSetDevice(device0);
    hipMemcpy(h_u, d0_u, size_u/2, hipMemcpyDeviceToHost);
    hipSetDevice(device1);
    hipMemcpy(h_u + size_u_p2, d1_u, size_u/2, hipMemcpyDeviceToHost);
    double time_IO_2 = omp_get_wtime() - time_tmp; 

    tot_time_compute += time_IO_1 + time_IO_2;

    // stats
    double GB = 1.0e-09;
    double flop = max_iter * (double)(N-2) * (double)(N-2) * 10.0;
    double gflops  = (flop / tot_time_compute) * GB;
    double memory  = size_f + size_u + size_u_old;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    printf("%g\t", memoryGBs); // bandwidth GB/s
    printf("%g\t", tot_time_compute); // total time
    printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    printf("%g\t", tot_time_compute); // compute time
    printf("# gpu3\n");

    //write_result(h_u, N, delta, "./../../analysis/pos/jac_gpu3.txt");

    // peer enable
    hipSetDevice(device0);
    hipDeviceDisablePeerAccess(device1);
    hipSetDevice(device1);
    hipDeviceDisablePeerAccess(device0);

    // free mem
    hipFree(d0_f), hipFree(d0_u), hipFree(d0_u_old);
    hipFree(d1_f), hipFree(d1_u), hipFree(d1_u_old);
    hipHostFree(h_f), hipHostFree(h_u), hipHostFree(h_u_old);
    // end program
    return(0);
}
