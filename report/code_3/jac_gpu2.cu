extern "C" { 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

void write_result(double *U, int N, double delta, char filename[40]) {
    double u, y, x;
    FILE *matrix=fopen(filename, "w");
    for (int i = 0; i < N; i++) {
        x = -1.0 + i * delta + delta * 0.5;
        for (int j = 0; j < N; j++) {
            y = -1.0 + j * delta + delta * 0.5;
            u = U[i*N + j];
            fprintf(matrix, "%g\t%g\t%g\n", x,y,u);
        }
    }
    fclose(matrix);
}
}

const int device0 = 0;
#define BLOCK_SIZE 16

void __global__ jac_gpu2(int N, double delta, int max_iter, double *f, double *u, double *u_old) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < (N-1) && j < (N-1) && i > 0 && j > 0) {
        u[i*N + j] = 0.25 * (u_old[(i-1)*N + j] + u_old[(i+1)*N + j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] + delta*delta*f[i*N + j]);
    }
}

int main(int argc, char *argv[]) {

    // warm up:
    double *dummy_d;
    hipSetDevice(device0);
    hipMalloc((void**)&dummy_d, 0);

    int max_iter, N,i,j;

    if (argc == 3) {
        N = atoi(argv[1]) + 2;
        max_iter = atoi(argv[2]);
    }
    else {
        // use default N
        N = 128 + 2;
        max_iter = 5000;
    }
    double delta = 2.0/N;

    // allocate mem
    double *h_f, *h_u, *h_u_old, *d_f, *d_u, *d_u_old;

    int size_f = N * N * sizeof(double);
    int size_u = N * N * sizeof(double);
    int size_u_old = N * N * sizeof(double);

    //Allocate memory on device
    hipSetDevice(device0);
    hipMalloc((void**)&d_f, size_f);
    hipMalloc((void**)&d_u, size_u);
    hipMalloc((void**)&d_u_old, size_u_old);
    //Allocate memory on host
    hipHostMalloc((void**)&h_f, size_f, hipHostMallocDefault);
    hipHostMalloc((void**)&h_u, size_u, hipHostMallocDefault);
    hipHostMalloc((void**)&h_u_old, size_u_old, hipHostMallocDefault);

    // initilize boarder
    for (i = 0; i < N; i++){
        for (j = 0; j < N; j++){
            if (i >= N * 0.5  &&  i <= N * 2.0/3.0  &&  j >= N * 1.0/6.0  &&  j <= N * 1.0/3.0)
                h_f[i*N + j] = 200.0;
            else
                h_f[i*N + j] = 0.0; 

            if (i == (N - 1) || i == 0 || j == (N - 1)){
                h_u[i*N + j] = 20.0;
                h_u_old[i*N + j] = 20.0;
            }
            else{
                h_u[i*N + j] = 0.0;
                h_u_old[i*N + j] = 0.0;
            } 
        }
    }
    
    //Copy memory host -> device
    double time_tmp = omp_get_wtime(); 
    hipMemcpy(d_f, h_f, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_u, h_u, size_u_old, hipMemcpyHostToDevice);
    hipMemcpy(d_u_old, h_u_old, size_u_old, hipMemcpyHostToDevice);
    double time_IO_1 = omp_get_wtime() - time_tmp; 

    // do program
    int k = 0;
    dim3 dim_grid(((N+BLOCK_SIZE-1) / BLOCK_SIZE), ((N+BLOCK_SIZE-1) / BLOCK_SIZE));
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    double *temp, time_compute = omp_get_wtime(); 
    while (k < max_iter) {
        // Set u_old = u
        temp = d_u;
        d_u = d_u_old;
        d_u_old = temp;
        jac_gpu2<<<dim_grid,dim_block>>>(N, delta, max_iter, d_f, d_u, d_u_old);
        hipDeviceSynchronize();
        k++;
    }/* end while */
    double tot_time_compute = omp_get_wtime() - time_compute;
    // end program

    //Copy memory host -> device
    time_tmp = omp_get_wtime(); 
    hipMemcpy(h_u, d_u, size_u, hipMemcpyDeviceToHost);
    double time_IO_2 = omp_get_wtime() - time_tmp; 

    tot_time_compute += time_IO_1 + time_IO_2;

    // stats
    double GB = 1.0e-09;
    double flop = max_iter * (double)(N-2) * (double)(N-2) * 10.0;
    double gflops  = (flop / tot_time_compute) * GB;
    double memory  = size_f + size_u + size_u_old;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    printf("%g\t", memoryGBs); // bandwidth GB/s
    printf("%g\t", tot_time_compute); // total time
    printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    printf("%g\t", tot_time_compute); // compute time
    printf("# gpu2\n");

    //write_result(h_u, N, delta, "./../../analysis/pos/jac_gpu2.txt");

    // free mem
    hipFree(d_f), hipFree(d_u), hipFree(d_u_old);
    hipHostFree(h_f), hipHostFree(h_u), hipHostFree(h_u_old);
    // end program
    return(0);
}
